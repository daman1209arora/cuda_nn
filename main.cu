#include "hip/hip_runtime.h"
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <algorithm>
#include "hipblas.h"
#include "layer.h"
#include "nn.h"


#define IDX2C(i,j,ld) (((j)*(ld))+(i))

struct Dataset {
    float* X;
    float* Y;
    int N, Fx, Fy;
};

Dataset* load_dataset(std::string filename, int lim=-1) {
    std::ifstream ifs(filename.c_str());

    std::string dataset_info;
    getline(ifs, dataset_info);
    std::stringstream ss(dataset_info);

    std::string tok;
    ss >> tok;
    int N;
    if(lim == -1)
        N = atoi(tok.c_str());
    else
        N = min(atoi(tok.c_str()), lim);
    
    ss >> tok;
    int Fx = atoi(tok.c_str());
    ss >> tok;
    int Fy = atoi(tok.c_str());

    float* X = (float*)malloc(N * Fx * sizeof(float));
    float* Y = (float*)malloc(N * Fy * sizeof(float));

    for(int i = 0; i < N; i++) {
        std::string s;
        getline(ifs, s);
        std::stringstream sX(s);
        for(int j = 0; j < Fx; j++) {
            std::string tok;
            sX >> tok;
            X[IDX2C(i, j, N)] = atof(tok.c_str());
        }
        
        getline(ifs, s);
        std::stringstream sY(s);
        for(int j = 0; j < Fy; j++) {
            std::string tok;
            sY >> tok;
            Y[IDX2C(i, j, N)] = atof(tok.c_str());
        }
    }

    Dataset* ds = (Dataset*)malloc(sizeof(Dataset));
    ds->X = X; ds->Y = Y; ds->N = N; ds->Fx = Fx; ds->Fy = Fy;
    
    return ds;
}

__global__ void addBias(float* WXdevPtr, float* bias, int N, int M) {
    // Adds bias to WXdevPtr
    int idx = blockIdx.x * gridDim.x + threadIdx.x;
    int biasIdx = (idx / (N / M));
    if((idx < N) && biasIdx < M) {
        WXdevPtr[idx] += bias[biasIdx];
    }
}

__global__ void relu(float* a, float* b, int N) {
    int idx = blockIdx.x * gridDim.x + threadIdx.x;
    if(idx < N) {
        if(a[idx] < 0)
            b[idx] = 0;
        else
            b[idx] = a[idx];
    }
}

int main(int argc, char* argv[]) {
    Dataset* ds = load_dataset("datasets/two-spiral.train", 5);
    
    int N = ds->N, Fx = ds->Fx, Fy = ds->Fy;
    float* X = ds->X;

    const int layerSizes[] = {3, 2};
    NN model = NN(layerSizes, Fx, Fy);
    
    float* XdevPtr = nullptr;
    
    hipMalloc((void**)&XdevPtr, N * Fx * sizeof(float));
    hipblasHandle_t handleX;
    hipblasCreate(&handleX);
    hipblasSetMatrix(N, Fx, sizeof(float), X, N, XdevPtr, N);
    
    
    /*hipFree(XdevPtr);
    hipblasDestroy(handleX);
    free(ds->X); 
    free(ds->Y);
    */
}