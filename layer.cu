#include "hip/hip_runtime.h"
#include "layer.h"
#include <cmath>
#include <random>
#include <stdlib.h>
#include <iostream>
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

float uniform_real(float l, float r) {
    float rand = random() / RAND_MAX;
    return rand * (r - l) + l;
}

Layer::Layer() {}

Layer::Layer(int R, int C) {

    std::random_device rd;  // Will be used to obtain a seed for the random number engine
    std::mt19937 gen(rd()); // S

    W = (float*)malloc(R * C * sizeof(float));
    b = (float*)malloc(C * sizeof(float));

    float std = 1.0f / sqrt(C);
    std::uniform_real_distribution<> dist(-std, std);

    for(int i = 0; i < R; i++) {
        for(int j = 0; j < C; j++)
            W[IDX2C(i, j, R)] = dist(gen);
    }

    for(int i = 0; i < C; i++) {
        b[IDX2C(i, 0, R)] = dist(gen);
    }
}

float* Layer::forward() {
    return NULL;
}